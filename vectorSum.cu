#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<stdio.h>


__global__ void vectorAdd(int* a, int* b, int* c) {
	//Lista de threads
	int i = threadIdx.x;

	c[i] = a[i] + b[i];

	return;

}

int main() {
	int a[] = { 1,2,3 };
	int b[] = { 4,5,6 };

	int c[sizeof(a) / sizeof(int)] = { 0 };


	// cria ponteiros para GPU
	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	//aloca memória na GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	// cópia os vetores da CPU para GPU
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);



	// GRID_SIZE,BLOCK_SIZE 
	vectorAdd << <1, sizeof(a) / sizeof(int) >> > (cudaA, cudaB, cudaC);

	//Copia da GPU para a CPU
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < sizeof(c) / sizeof(int); ++i)
		printf("c[%d] = %d\n", i, c[i]);

	return;
}
